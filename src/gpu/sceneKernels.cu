#include "hip/hip_runtime.h"
// Copyright 2024 Robotec.AI
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gpu/kernelUtils.hpp>
#include <gpu/sceneKernels.hpp>

__global__ void kPerformSkeletonAnimation(size_t vertexCount, const Vec3f* restposeVertices, const BoneWeights* boneWeights,
                                          const Mat3x4f* animationMatrices, Vec3f* skinnedVertices)
{
	LIMIT(vertexCount);

	skinnedVertices[tid] = (animationMatrices[boneWeights[tid].boneIndexes.x] * restposeVertices[tid]) *
	                       boneWeights[tid].weights.x;
	skinnedVertices[tid] += (animationMatrices[boneWeights[tid].boneIndexes.y] * restposeVertices[tid]) *
	                        boneWeights[tid].weights.y;
	skinnedVertices[tid] += (animationMatrices[boneWeights[tid].boneIndexes.z] * restposeVertices[tid]) *
	                        boneWeights[tid].weights.z;
	skinnedVertices[tid] += (animationMatrices[boneWeights[tid].boneIndexes.w] * restposeVertices[tid]) *
	                        boneWeights[tid].weights.w;
}

__global__ void kCalculateAnimationMatrices(size_t boneCount, const Mat3x4f* restposes, Mat3x4f* animationMatrices)
{
	LIMIT(boneCount);
	animationMatrices[tid] = animationMatrices[tid] * restposes[tid];
}

// Updates vertices and calculates their displacement.
// Input: newVertices and oldVertices
// Output: verticesDisplacement and newVertices
__global__ void kUpdateVertices(size_t vertexCount, Vec3f* newVerticesToDisplacement, Vec3f* oldToNewVertices)
{
	LIMIT(vertexCount);
	// See ExternalAnimator::animate or SkeletonAnimator::animate to understand the logic here.
	Vec3f newVertex = newVerticesToDisplacement[tid];
	newVerticesToDisplacement[tid] -= oldToNewVertices[tid];
	oldToNewVertices[tid] = newVertex;
}

void gpuPerformSkeletonAnimation(hipStream_t stream, size_t vertexCount, size_t boneCount, const Vec3f* restposeVertices,
                                 const BoneWeights* boneWeights, const Mat3x4f* restposes, Mat3x4f* animationMatrices,
                                 Vec3f* skinnedVertices)
{
	run(kCalculateAnimationMatrices, stream, boneCount, restposes, animationMatrices);
	run(kPerformSkeletonAnimation, stream, vertexCount, restposeVertices, boneWeights, animationMatrices, skinnedVertices);
}

void gpuUpdateVerticesWithDisplacement(hipStream_t stream, size_t vertexCount, Vec3f* newVerticesToDisplacement,
                                       Vec3f* oldToNewVertices)
{
	run(kUpdateVertices, stream, vertexCount, newVerticesToDisplacement, oldToNewVertices);
}
