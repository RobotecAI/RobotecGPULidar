#include "hip/hip_runtime.h"
// Copyright 2022 Robotec.AI
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gpu/kernelUtils.hpp>
#include <gpu/nodeKernels.hpp>
#include <gpu/GPUFieldDesc.hpp>
#include <macros/cuda.hpp>
#include <vector>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

__global__ void kFormatSoaToAos(size_t pointCount, size_t pointSize, size_t fieldCount, const GPUFieldDesc *soaInData, char *aosOutData)
{
	LIMIT(pointCount);
	// Implement padding
	for (size_t i = 0; i < fieldCount; ++i) {
		memcpy(aosOutData + pointSize * tid + soaInData[i].dstOffset, soaInData[i].readDataPtr + soaInData[i].size * tid, soaInData[i].size);
	}
}

__global__ void kFormatAosToSoa(size_t pointCount, size_t pointSize, size_t fieldCount, const char* aosInData, GPUFieldDesc* soaOutData)
{
	LIMIT(pointCount);
	// Implement padding
	for (size_t i = 0; i < fieldCount; ++i) {
		memcpy(soaOutData[i].writeDataPtr + soaOutData[i].size * tid, aosInData + pointSize * tid + soaOutData[i].dstOffset, soaOutData[i].size);
	}
}

__global__ void kTransformRays(size_t rayCount, const Mat3x4f* inRays, Mat3x4f* outRays, Mat3x4f transform)
{
	LIMIT(rayCount);
	outRays[tid] = transform * inRays[tid];
}

__global__ void kTransformPoints(size_t pointCount, const Field<XYZ_F32>::type* inPoints, Field<XYZ_F32>::type* outPoints, Mat3x4f transform)
{
	LIMIT(pointCount);
	outPoints[tid] = transform * inPoints[tid];
}

__global__ void kApplyCompaction(size_t pointCount, size_t fieldSize, const Field<IS_HIT_I32>::type* shouldWrite, const CompactionIndexType*writeIndex, char *dst, const char *src)
{
	LIMIT(pointCount);
	int32_t rIdx = tid;
	if (!shouldWrite[rIdx]) {
		return;
	}
	int wIdx = writeIndex[rIdx] - 1;
	memcpy(dst + fieldSize * wIdx, src + fieldSize * rIdx, fieldSize);
}

__global__ void kCutField(size_t pointCount, char* dst, const char* src, size_t offset, size_t stride, size_t fieldSize)
{
	LIMIT(pointCount);
	memcpy(dst + tid * fieldSize, src + tid * stride + offset, fieldSize);
}

__global__ void kFilter(size_t count, const Field<RAY_IDX_U32>::type* indices, char* dst, char* src, size_t fieldSize)
{
	LIMIT(count);
	memcpy(dst + tid * fieldSize, src + indices[tid] * fieldSize, fieldSize);
}

void gpuFindCompaction(hipStream_t stream, size_t pointCount, const Field<IS_HIT_I32>::type* isHit, CompactionIndexType* hitCountInclusive, size_t* outHitCount)
{
	// beg and end could be used as const pointers, however thrust does not support it
	auto beg = thrust::device_ptr<const int32_t>(isHit);
	auto end = thrust::device_ptr<const int32_t>(isHit + pointCount);
	auto dst = thrust::device_ptr<int32_t>(hitCountInclusive);

	// Note: this will compile only in a .cu file
	thrust::inclusive_scan(thrust::cuda::par.on(stream), beg, end, dst);
	CHECK_CUDA(hipMemcpyAsync(outHitCount, hitCountInclusive + pointCount - 1, sizeof(*hitCountInclusive), hipMemcpyDefault, stream));
}

void gpuFormatSoaToAos(hipStream_t stream, size_t pointCount, size_t pointSize, size_t fieldCount, const GPUFieldDesc *soaInData, char *aosOutData)
{ run(kFormatSoaToAos, stream, pointCount, pointSize, fieldCount, soaInData, aosOutData); }

void gpuFormatAosToSoa(hipStream_t stream, size_t pointCount, size_t pointSize, size_t fieldCount, const char* aosInData, GPUFieldDesc* soaOutData)
{ run(kFormatAosToSoa, stream, pointCount, pointSize, fieldCount, aosInData, soaOutData); }

void gpuTransformRays(hipStream_t stream, size_t rayCount, const Mat3x4f* inRays, Mat3x4f* outRays, Mat3x4f transform)
{ run(kTransformRays, stream, rayCount, inRays, outRays, transform); };

void gpuApplyCompaction(hipStream_t stream, size_t pointCount, size_t fieldSize, const Field<IS_HIT_I32>::type* shouldWrite, const CompactionIndexType *writeIndex, char *dst, const char *src)
{ run(kApplyCompaction, stream, pointCount, fieldSize, shouldWrite, writeIndex, dst, src); }

void gpuTransformPoints(hipStream_t stream, size_t pointCount, const Field<XYZ_F32>::type* inPoints, Field<XYZ_F32>::type* outPoints, Mat3x4f transform)
{ run(kTransformPoints, stream, pointCount, inPoints, outPoints, transform); }

void gpuCutField(hipStream_t stream, size_t pointCount, char *dst, const char *src, size_t offset, size_t stride, size_t fieldSize)
{ run(kCutField, stream, pointCount, dst, src, offset, stride, fieldSize); }

void gpuFilter(hipStream_t stream, size_t count, const Field<RAY_IDX_U32>::type* indices, char *dst, const char *src, size_t fieldSize)
{ run(kFilter, stream, count, indices, dst, src, fieldSize); }
