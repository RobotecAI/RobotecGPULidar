#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <optix_device.h>

#include <math/Vector.hpp>
#include <math/Mat3x4f.hpp>
#include <cassert>

#include <gpu/RaytraceRequestContext.hpp>
#include <gpu/ShaderBindingTableTypes.h>


extern "C" static __constant__ RaytraceRequestContext ctx;

template<bool isFinite>
__forceinline__ __device__
void saveRayResult(Vec3f* xyz=nullptr)
{
	const int rayIdx = optixGetLaunchIndex().x;

	if (ctx.xyz != nullptr) {
		// Return actual XYZ of the hit point or infinity vector with signs of the ray.
		ctx.xyz[rayIdx] = isFinite ? *xyz : ctx.rays[rayIdx] * Vec3f{HIP_INF_F, HIP_INF_F, HIP_INF_F};
	}
	if (ctx.isHit != nullptr) {
		ctx.isHit[rayIdx] = isFinite;
	}
	if (ctx.rayIdx != nullptr) {
		ctx.rayIdx[rayIdx] = rayIdx;
	}
}

extern "C" __global__ void __raygen__()
{
	Mat3x4f ray = ctx.rays[optixGetLaunchIndex().x];

	Vec3f origin = ray * Vec3f{0, 0, 0};
	Vec3f dir = ray * Vec3f{0, 0, 1} - origin;

	unsigned int flags = OPTIX_RAY_FLAG_DISABLE_ANYHIT;
	optixTrace(ctx.scene, origin, dir, 0.0f, ctx.rayRange, 0.0f, OptixVisibilityMask(255), flags, 0, 1, 0);
}

extern "C" __global__ void __closesthit__()
{
	const TriangleMeshSBTData& sbtData
		= *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

	const int primID = optixGetPrimitiveIndex();
	assert(primID < sbtData.index_count);
	const Vec3i index = sbtData.index[primID];
	const float u = optixGetTriangleBarycentrics().x;
	const float v = optixGetTriangleBarycentrics().y;

	assert(index.x() < sbtData.vertex_count);
	assert(index.y() < sbtData.vertex_count);
	assert(index.z() < sbtData.vertex_count);
	const Vec3f& A = sbtData.vertex[index.x()];
	const Vec3f& B = sbtData.vertex[index.y()];
	const Vec3f& C = sbtData.vertex[index.z()];

	Vec3f hitObject = Vec3f((1 - u - v) * A + u * B + v * C);
	Vec3f hitWorld = optixTransformPointFromObjectToWorldSpace(hitObject);

	saveRayResult<true>(&hitWorld);
}

extern "C" __global__ void __miss__()
{
	saveRayResult<false>();
}

extern "C" __global__ void __anyhit__(){}
