#include "hip/hip_runtime.h"
// Copyright 2023 Robotec.AI
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <gpu/kernelUtils.hpp>
#include <gpu/helpersKernels.hpp>

// Philox algorithm chosen based on performance
// https://stackoverflow.com/questions/18506697/hiprand-properties-of-generators
__global__ void kSetupRandomNumberGenerator(size_t pointCount, unsigned int seed, hiprandStatePhilox4_32_10_t* states)
{
	LIMIT(pointCount);
	/* Each thread gets same seed, a different sequence number, no offset */
	hiprand_init(seed, tid, 0, &states[tid]);
}

void gpuSetupRandomNumberGenerator(hipStream_t stream, size_t elementsCount, unsigned int seed,
                                   hiprandStatePhilox4_32_10_t* outPHILOXStates)
{
	run(kSetupRandomNumberGenerator, stream, elementsCount, seed, outPHILOXStates);
}
// Updates vertices and calculates their displacement.
// Input: newVertices and oldVertices
// Output: verticesDisplacement and newVertices
__global__ void kUpdateVertices(size_t vertexCount, Vec3f* newVerticesToDisplacement, Vec3f* oldToNewVertices)
{
	LIMIT(vertexCount);
	// See Mesh::updateVertices to understand the logic here.
	Vec3f newVertex = newVerticesToVelocity[tid];
	newVerticesToVelocity[tid] -= oldToNewVertices[tid];
	oldToNewVertices[tid] = newVertex;
}

void gpuUpdateVertices(hipStream_t stream, size_t vertexCount, Vec3f* newVerticesToVelocity, Vec3f* oldToNewVertices)
{
	run(kUpdateVertices, stream, vertexCount, newVerticesToVelocity, oldToNewVertices);
}
