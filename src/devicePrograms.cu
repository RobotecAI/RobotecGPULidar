#include "hip/hip_runtime.h"
#include "LaunchParams.h"
#include "data_types/ShaderBindingTableTypes.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>

#define HOSTDEVICE __device__
#include "linearGeometry.h"

#define NDEBUG
//#undef NDEBUG
#include <assert.h>

extern "C" static __constant__ LaunchLidarParams optixLaunchLidarParams;

static __forceinline__ __device__ void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

extern "C" __global__ void __closesthit__lidar()
{
    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    const int primID = optixGetPrimitiveIndex();
    assert(primID < sbtData.index_count);
    const Vec3i index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    assert(index.x() < sbtData.vertex_count);
    assert(index.y() < sbtData.vertex_count);
    assert(index.z() < sbtData.vertex_count);
    const Vec3f& A = sbtData.vertex[index.x()];
    const Vec3f& B = sbtData.vertex[index.y()];
    const Vec3f& C = sbtData.vertex[index.z()];

    Vec3f& prd = *(Vec3f*)getPRD<Vec3f>();
    prd = Vec3f((1 - u - v) * A + u * B + v * C);

    gdt::vec3f rayHitPoint = gdt::vec3f(prd.x(), prd.y(), prd.z());
    gdt::vec3f unityPoint = optixTransformPointFromObjectToWorldSpace(rayHitPoint);
    gdt::vec3f rosPoint = multiply3x4TransformByVector3(optixLaunchLidarParams.rosTransform, unityPoint);

    const int ix = optixGetLaunchIndex().x;

    optixLaunchLidarParams.dUnityVisualisationPoints[ix].x = unityPoint.x;
    optixLaunchLidarParams.dUnityVisualisationPoints[ix].y = unityPoint.y;
    optixLaunchLidarParams.dUnityVisualisationPoints[ix].z = unityPoint.z;
    optixLaunchLidarParams.dRosXYZ[ix].x = rosPoint.x;
    optixLaunchLidarParams.dRosXYZ[ix].y = rosPoint.y;
    optixLaunchLidarParams.dRosXYZ[ix].z = rosPoint.z;

    optixSetPayload_3(1);
}

extern "C" __global__ void __anyhit__lidar()
{ /*! for this simple example, this will remain empty */
}

extern "C" __global__ void __miss__lidar()
{
    gdt::vec3f& prd = *(gdt::vec3f*)getPRD<gdt::vec3f>();
    prd = gdt::vec3f(0.f);
    optixSetPayload_3(0);
}

extern "C" __global__ void __raygen__renderLidar()
{
    const int ix = optixGetLaunchIndex().x;

    gdt::vec3f lidarPositionPRD = gdt::vec3f(0.f);

    // the values we store the PRD pointer in:
    uint32_t u0 = 0U;
    uint32_t u1 = 0U;
    uint32_t u2 = 0U;
    uint32_t u3 = 0U;
    packPointer(&lidarPositionPRD, u0, u1);

    TransformMatrix ray_pose_local = optixLaunchLidarParams.dRayPoses[ix];
    TransformMatrix ray_pose_global = multiply3x4TransformMatrices(optixLaunchLidarParams.lidarPose, ray_pose_local);

    gdt::vec3f from = getTranslationFrom3x4Transform(ray_pose_global);
    gdt::vec3f zero = gdt::vec3f(0.0f, 0.0f, 0.0f);
    gdt::vec3f forward = gdt::vec3f(0.0f, 0.0f, 1.0f);
    gdt::vec3f zero_moved = multiply3x4TransformByVector3(ray_pose_global, zero);
    gdt::vec3f forward_moved = multiply3x4TransformByVector3(ray_pose_global, forward);
    gdt::vec3f dir = gdt::vec3f(forward_moved.x - zero_moved.x, forward_moved.y - zero_moved.y, forward_moved.z - zero_moved.z) ;

    optixTrace(optixLaunchLidarParams.traversable,
        from,
        dir,
        0.f, // tmin
        optixLaunchLidarParams.range,
        0.0f, // rayTime
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
        LIDAR_RAY_TYPE, // SBT offset
        LIDAR_RAY_TYPE_COUNT, // SBT stride
        LIDAR_RAY_TYPE, // missSBTIndex
        u0, u1, u2, u3);

    if (u3) {
        optixLaunchLidarParams.dWasHit[ix] = 1;
    } else {
        optixLaunchLidarParams.dWasHit[ix] = 0;
    }
}
