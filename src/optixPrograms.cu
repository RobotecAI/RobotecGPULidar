#include "hip/hip_runtime.h"
#include "LaunchParams.h"
#include "data_types/ShaderBindingTableTypes.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <math/Vector.hpp>

#define HOSTDEVICE __device__
#include "linearGeometry.h"


extern "C" static __constant__ LaunchLidarParams optixLaunchLidarParams;

static __forceinline__ __device__ void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

extern "C" __global__ void __closesthit__lidar()
{
    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    const int primID = optixGetPrimitiveIndex();
    assert(primID < sbtData.index_count);
    const Vec3i index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    assert(index.x() < sbtData.vertex_count);
    assert(index.y() < sbtData.vertex_count);
    assert(index.z() < sbtData.vertex_count);
    const Vec3f& A = sbtData.vertex[index.x()];
    const Vec3f& B = sbtData.vertex[index.y()];
    const Vec3f& C = sbtData.vertex[index.z()];

    Vec3f& prd = *(Vec3f*)getPRD<Vec3f>();
    prd = Vec3f((1 - u - v) * A + u * B + v * C);

    Vec3f rayHitPoint = Vec3f(prd.x(), prd.y(), prd.z());
    Vec3f unityPoint = optixTransformPointFromObjectToWorldSpace(rayHitPoint);
    Vec3f rosPoint = multiply3x4TransformByVector3(optixLaunchLidarParams.rosTransform, unityPoint);

    const int ix = optixGetLaunchIndex().x;

    optixLaunchLidarParams.dUnityVisualisationPoints[ix].x = unityPoint.x();
    optixLaunchLidarParams.dUnityVisualisationPoints[ix].y = unityPoint.y();
    optixLaunchLidarParams.dUnityVisualisationPoints[ix].z = unityPoint.z();
    optixLaunchLidarParams.dRosXYZ[ix].x = rosPoint.x();
    optixLaunchLidarParams.dRosXYZ[ix].y = rosPoint.y();
    optixLaunchLidarParams.dRosXYZ[ix].z = rosPoint.z();

    optixSetPayload_3(1);
}

extern "C" __global__ void __anyhit__lidar()
{ /*! for this simple example, this will remain empty */
}

extern "C" __global__ void __miss__lidar()
{
    Vec3f& prd = *(Vec3f*)getPRD<Vec3f>();
    prd = Vec3f(0.f);
    optixSetPayload_3(0);
}

extern "C" __global__ void __raygen__renderLidar()
{
    const int ix = optixGetLaunchIndex().x;

    Vec3f lidarPositionPRD = Vec3f(0.f);

    // the values we store the PRD pointer in:
    uint32_t u0 = 0U;
    uint32_t u1 = 0U;
    uint32_t u2 = 0U;
    uint32_t u3 = 0U;
    packPointer(&lidarPositionPRD, u0, u1);

    Mat3x4f ray_pose_local = optixLaunchLidarParams.dRayPoses[ix];
    Mat3x4f ray_pose_global = multiply3x4TransformMatrices(optixLaunchLidarParams.lidarPose, ray_pose_local);

    Vec3f from = getTranslationFrom3x4Transform(ray_pose_global);
    Vec3f zero = Vec3f(0.0f, 0.0f, 0.0f);
    Vec3f forward = Vec3f(0.0f, 0.0f, 1.0f);
    Vec3f zero_moved = multiply3x4TransformByVector3(ray_pose_global, zero);
    Vec3f forward_moved = multiply3x4TransformByVector3(ray_pose_global, forward);
    Vec3f dir = Vec3f(forward_moved.x() - zero_moved.x(), forward_moved.y() - zero_moved.y(), forward_moved.z() - zero_moved.z()) ;

    optixTrace(optixLaunchLidarParams.traversable,
        from,
        dir,
        0.f, // tmin
        optixLaunchLidarParams.range,
        0.0f, // rayTime
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
        LIDAR_RAY_TYPE, // SBT offset
        LIDAR_RAY_TYPE_COUNT, // SBT stride
        LIDAR_RAY_TYPE, // missSBTIndex
        u0, u1, u2, u3);

    if (u3) {
        optixLaunchLidarParams.dWasHit[ix] = 1;
    } else {
        optixLaunchLidarParams.dWasHit[ix] = 0;
    }
}
